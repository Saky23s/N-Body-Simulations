#include "hip/hip_runtime.h"

/** 
 * @file metodo_directo_grav.c
 * @author Santiago Salas santiago.salas@estudiante.uam.es
 * 
 * File that does the operations for the acceletation of each body suffered by the effects of the gravitational pull of all other bodies
 * 
 * This is the point where most of the time of the simulation happens, optimizing this file even by a little has grat impact on the 
 * total performance
 * 
 * In this case we are compiling this file with -O3 for maximum optimizations, this does not affect results but speeds performance
 */

#include "../inc/medoto_directo_defs.h"

//Macros to correctly access multidimensional arrays that has been flatten 
#define S(size_i, size_j, cord, i, j, pointer) pointer[(size_i * size_j * cord) + (size_j * i) + j]

//Cuda kernels
__device__ void calculate_acceleration_values(realptr d_masses, realptr position, realptr sdata, int n);

template <unsigned int blockSize>
__device__ void warpReduce(volatile realptr sdata, int baseIndex0, int baseIndex1, int baseIndex2);

template <unsigned int blockSize>
__global__ void calculate_acceleration_values_block_reduce(realptr d_masses, realptr position, realptr d_block_holder, int n, unsigned int number_of_blocks_j);

template <unsigned int blockSize>
__device__ void full_block_reduction (realptr d_block_holder, realptr sdata, int n, unsigned int number_of_blocks_j);

__global__ void finish_block_reduce (realptr acceleration, realptr d_block_holder, int n, unsigned int number_of_blocks_j);

#define cudaErrorCheck(ans) { gpuAssert((ans), __FILE__, __LINE__); }

/**
 * Function to check errors in CUDA. 
 * 
 * Extracted from StackOverflow 
 * @link https://stackoverflow.com/a/14038590
 */
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
    if (code != hipSuccess) 
    {
        fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
        if (abort) exit(code);
    }
}

int calculate_acceleration(Simulation* simulation)
/**
 * Funtion to calculate the velocity and acceleration of the bodies using the current positions and velocities. 
 * It uses a cuda kernel to do GPU acceleration and calculate the values
 * 
 * @param simulation(Simulation*): a pointer to the simulation object we are simulating
 * 
 * @return status (int): STATUS_ERROR (0) in case of error STATUS_OK(1) in case everything when ok
 *         The resulting acceleration is stored inside acceleration atribute of the simulation
**/
{   
    //Error checking
    if(simulation == NULL)
        return STATUS_ERROR;

    //Set up memory for cuda as 0 for reduccions
    hipMemset( simulation->d_block_holder, 0.0, 3 * simulation->n  * simulation->gridDimsGrav.y * sizeof(simulation->d_block_holder[0]));
    hipMemset( simulation->d_acceleration, 0.0, 3 * simulation->n  * sizeof(simulation->d_acceleration[0]));

    //Call cuda with the correct block size
    switch (simulation->threadBlockDimsGrav.y)
    {   
        case 1024:
            calculate_acceleration_values_block_reduce<1024><<<simulation->gridDimsGrav, simulation->threadBlockDimsGrav, 3 * simulation->threadBlockDimsGrav.x * simulation->threadBlockDimsGrav.y * sizeof(real)>>>(simulation->d_masses, simulation->d_positions, simulation->d_block_holder ,simulation->n, simulation->gridDimsGrav.y);
            break;
        case 512:
            calculate_acceleration_values_block_reduce<512><<<simulation->gridDimsGrav, simulation->threadBlockDimsGrav, 3 * simulation->threadBlockDimsGrav.x * simulation->threadBlockDimsGrav.y * sizeof(real)>>>(simulation->d_masses, simulation->d_positions, simulation->d_block_holder ,simulation->n, simulation->gridDimsGrav.y);
            break;
        case 256:
            calculate_acceleration_values_block_reduce<256><<<simulation->gridDimsGrav, simulation->threadBlockDimsGrav, 3 * simulation->threadBlockDimsGrav.x * simulation->threadBlockDimsGrav.y * sizeof(real)>>>(simulation->d_masses, simulation->d_positions, simulation->d_block_holder ,simulation->n, simulation->gridDimsGrav.y);
            break;
        case 128:
            calculate_acceleration_values_block_reduce<128><<<simulation->gridDimsGrav, simulation->threadBlockDimsGrav, 3 * simulation->threadBlockDimsGrav.x * simulation->threadBlockDimsGrav.y * sizeof(real)>>>(simulation->d_masses, simulation->d_positions, simulation->d_block_holder ,simulation->n, simulation->gridDimsGrav.y);
            break;
        case 64:
            calculate_acceleration_values_block_reduce<64><<<simulation->gridDimsGrav, simulation->threadBlockDimsGrav, 3 * simulation->threadBlockDimsGrav.x * simulation->threadBlockDimsGrav.y * sizeof(real)>>>(simulation->d_masses, simulation->d_positions, simulation->d_block_holder ,simulation->n, simulation->gridDimsGrav.y);
            break;            
        case 32:
            calculate_acceleration_values_block_reduce<32><<<simulation->gridDimsGrav, simulation->threadBlockDimsGrav, 3 * simulation->threadBlockDimsGrav.x * simulation->threadBlockDimsGrav.y * sizeof(real)>>>(simulation->d_masses, simulation->d_positions, simulation->d_block_holder ,simulation->n, simulation->gridDimsGrav.y);
            break;
    }
    
    
    hipError_t status = hipGetLastError();
    cudaErrorCheck(status);

    finish_block_reduce<<<simulation->gridDimsLeap,simulation->threadBlockLeap>>>(simulation->d_acceleration, simulation->d_block_holder, simulation->n, simulation->gridDimsGrav.y);
    
    status = hipGetLastError();
    cudaErrorCheck(status);
    return STATUS_OK;
}

template <unsigned int blockSize>
__global__ void calculate_acceleration_values_block_reduce(realptr d_masses, realptr position, realptr d_block_holder, int n, unsigned int number_of_blocks_j)
/**
 * A cuda kernel that calculate the 3n**2 acceleration values and reduce them to an array of size of 3n * gridDim.y. 
 * @param d_masses (realptr): A cuda array of size n with the masses of each body
 * @param d_position (realptr): A cuda array of size 3n with the current position of all of the bodies stored in the following matter x1,y1,z1,x2,y2,z2...xn,yn,zn
 * @param d_block_holder (realptr): An array where the block-reduced results will be stored. It must be of size 3n * gridDim.y
 * @param n (int): the number of bodies
 * @param number_of_blocks_j (unsigned int): The number of blocks in the j dimension. Used for the block reduction
 */
{   
    //Array where all values of this block will be stored
    extern __shared__ real sdata[];

    //Calculate aceleration values
    calculate_acceleration_values(d_masses, position, sdata, n);
        
    //Reduce all values of this block
    full_block_reduction<blockSize>(d_block_holder, sdata, n, number_of_blocks_j);
}


__device__ void calculate_acceleration_values(realptr d_masses, realptr position, realptr sdata, int n)
/**
 * Cuda kernel that calculates the acceleration values that each body suffers from every other body
 * @param d_masses (realptr): A cuda array of size n with the masses of each body
 * @param d_position (realptr): A cuda array of size 3n with the current position of all of the bodies stored in the following matter x1,y1,z1,x2,y2,z2...xn,yn,zn
 * @param sdata (realptr): A shared array in which to store all of the acceleration values
 * @param n (int): the number of bodies
 */
{
    //Get position in the block
    int x = threadIdx.x;
    int y = threadIdx.y;

    //Get universal position
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    int j = threadIdx.y + blockIdx.y * blockDim.y;    
    
    real softening2 = softening * softening;

    //Calculate pull of one body by other body
    if(i < n && j < n && i != j)
    {   
        real dx = position[j] - position[i]; //rx body 2 - rx body 1
        real dy = position[j + n] - position[i + n]; //ry body 2 - ry body 1
        real dz = position[j + n + n] - position[i + n + n]; //rz body 2 - rz body 1
        
        real r = rsqrt(dx * dx + dy * dy + dz * dz + softening2); //distance magnitud with some softening
        r = (G * d_masses[j] * r * r * r ); //Acceleration formula

        S(blockDim.x, blockDim.y, 0, x, y, sdata) =  r * dx; //Acceleration formula for x
        S(blockDim.x, blockDim.y, 1, x, y, sdata) =  r * dy; //Acceleration formula for y
        S(blockDim.x, blockDim.y, 2, x, y, sdata) =  r * dz; //Acceleration formula for z
    }
    //Fill with 0 the remaining values in the array with 0
    else
    {  
        S(blockDim.x, blockDim.y, 0, x, y, sdata) = 0.0;    //x
        S(blockDim.x, blockDim.y, 1, x, y, sdata) = 0.0;    //y
        S(blockDim.x, blockDim.y, 2, x, y, sdata) = 0.0;    //z
    }
    
}

template <unsigned int blockSize>
__device__ void full_block_reduction (realptr d_block_holder, realptr sdata, int n, unsigned int number_of_blocks_j)
/**
 * A cuda kernel that reduces the acceleration values of this block to one for every body in this block. 
 * It implements a modified version of reduction6 in https://developer.download.nvidia.com/assets/cuda/files/reduction.pdf
 * 
 * @param d_block_holder (realptr): An array where the block-reduced results will be stored. It must be of size 3n * gridDim.y
 * @param sdata (realptr): A shared array in which all of the aceleration values of this block are stored
 * @param n (int): the number of bodies
 * @param number_of_blocks_j (unsigned int): The number of blocks in the j dimension.
 */
{
    //Get position in the block
    int x = threadIdx.x;
    int y = threadIdx.y;

    //Get universal position
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    int baseIndex0 = (blockDim.y * x) + y;
    int baseIndex1 = (blockDim.x * blockDim.y) + (blockDim.y * x) + y;
    int baseIndex2 = (blockDim.x * blockDim.y * 2) + (blockDim.y * x) + y;

    __syncthreads();
    
    //Unrolled reduction
    if (blockSize >= 1024) 
    {   
        if (y < 512) 
        { 
            sdata[baseIndex0] += sdata[baseIndex0 + 512];
            sdata[baseIndex1] += sdata[baseIndex1 + 512];
            sdata[baseIndex2] += sdata[baseIndex2 + 512];
        }
        __syncthreads();
    }

    if (blockSize >= 512) 
    {   
        if (y < 256) 
        { 
            sdata[baseIndex0] += sdata[baseIndex0 + 256];
            sdata[baseIndex1] += sdata[baseIndex1 + 256];
            sdata[baseIndex2] += sdata[baseIndex2 + 256];
        }
        __syncthreads();
    }

    if (blockSize >= 256) 
    {   
        if (y < 128) 
        { 
            sdata[baseIndex0] += sdata[baseIndex0 + 128];
            sdata[baseIndex1] += sdata[baseIndex1 + 128];
            sdata[baseIndex2] += sdata[baseIndex2 + 128];
        }
        __syncthreads();
    }

    if (blockSize >= 128) 
    {   
        if (y < 64) 
        { 
            sdata[baseIndex0] += sdata[baseIndex0 + 64];
            sdata[baseIndex1] += sdata[baseIndex1 + 64];
            sdata[baseIndex2] += sdata[baseIndex2 + 64];
        }
        __syncthreads();
    }

    if (y < 32 && i < n) 
        warpReduce<blockSize>(sdata, baseIndex0, baseIndex1, baseIndex2);
    
    // write result for this block to global mem
    if (y == 0)
    {
        S(n, number_of_blocks_j, 0, i, blockIdx.y, d_block_holder) += sdata[baseIndex0];
        S(n, number_of_blocks_j, 1, i, blockIdx.y, d_block_holder) += sdata[baseIndex1];
        S(n, number_of_blocks_j, 2, i, blockIdx.y, d_block_holder) += sdata[baseIndex2];
    } 
}

template <unsigned int blockSize>
__device__ void warpReduce(volatile realptr sdata, int baseIndex0, int baseIndex1, int baseIndex2)
/**
 * A cuda kernel that helps in the reduction process. It completly reduces a warp
 * It implements a modified version of warpReduce from https://developer.download.nvidia.com/assets/cuda/files/reduction.pdf
 * 
 * @param sdata (realptr): A shared array in which all of the aceleration values of this block are stored
 * @param x (int): the x position in the block
 * @param y (int): the y position in the block
 */
{   
    if (blockSize >= 64) 
    {
        sdata[baseIndex0] += sdata[baseIndex0 + 32];
        sdata[baseIndex1] += sdata[baseIndex1 + 32];
        sdata[baseIndex2] += sdata[baseIndex2 + 32];
    }

    if (blockSize >= 32) 
    {
        sdata[baseIndex0] += sdata[baseIndex0 + 16];
        sdata[baseIndex1] += sdata[baseIndex1 + 16];
        sdata[baseIndex2] += sdata[baseIndex2 + 16];
    }

    if (blockSize >= 16)
    {
        sdata[baseIndex0] += sdata[baseIndex0 + 8];
        sdata[baseIndex1] += sdata[baseIndex1 + 8];
        sdata[baseIndex2] += sdata[baseIndex2 + 8];
    }   

    if (blockSize >= 8) 
    {
        sdata[baseIndex0] += sdata[baseIndex0 + 4];
        sdata[baseIndex1] += sdata[baseIndex1 + 4];
        sdata[baseIndex2] += sdata[baseIndex2 + 4];
    }

    if (blockSize >= 4) 
    {
        sdata[baseIndex0] += sdata[baseIndex0 + 2];
        sdata[baseIndex1] += sdata[baseIndex1 + 2];
        sdata[baseIndex2] += sdata[baseIndex2 + 2];
    }
    if (blockSize >= 2) 
    {
        sdata[baseIndex0] += sdata[baseIndex0 + 1];
        sdata[baseIndex1] += sdata[baseIndex1 + 1];
        sdata[baseIndex2] += sdata[baseIndex2 + 1];
    }
}

__global__ void finish_block_reduce (realptr acceleration, realptr d_block_holder, int n, unsigned int number_of_blocks_j)
{
    int i = threadIdx.x + blockIdx.x * blockDim.x;

    if (i < n) 
    {
        // Use registers to store intermediate sums
        real sum_x = 0.0;
        real sum_y = 0.0;
        real sum_z = 0.0;

        // Precompute base index part to avoid redundant calculation
        int baseIndex0 = number_of_blocks_j * i;  
        int baseIndex1 = n * number_of_blocks_j + number_of_blocks_j * i;  
        int baseIndex2 = n * number_of_blocks_j * 2 + number_of_blocks_j * i;  

        // Perform the reduction in the loop
        // Unrolling if number_of_blocks_j is small, this case for 4
        for (int j = 0; j < number_of_blocks_j; j += 4) 
        {
            sum_x += d_block_holder[baseIndex0 + j];
            sum_y += d_block_holder[baseIndex1 + j];
            sum_z += d_block_holder[baseIndex2 + j];

            //All threads should enter or not enter this loop
            if (j + 1 < number_of_blocks_j) 
            {
                sum_x += d_block_holder[baseIndex0 + j + 1];
                sum_y += d_block_holder[baseIndex1 + j + 1];
                sum_z += d_block_holder[baseIndex2 + j + 1];
            }
            if (j + 2 < number_of_blocks_j) 
            {
                sum_x += d_block_holder[baseIndex0 + j + 2];
                sum_y += d_block_holder[baseIndex1 + j + 2];
                sum_z += d_block_holder[baseIndex2 + j + 2];
            }
            if (j + 3 < number_of_blocks_j) 
            {
                sum_x += d_block_holder[baseIndex0 + j + 3];
                sum_y += d_block_holder[baseIndex1 + j + 3];
                sum_z += d_block_holder[baseIndex2 + j + 3];
            }
        }


        // After the loop, store the results to global memory
        acceleration[i] += sum_x;
        acceleration[i + n] += sum_y;
        acceleration[i + 2 * n] += sum_z;
    }
}

