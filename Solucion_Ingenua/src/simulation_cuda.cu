#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <sys/time.h>
#include "../inc/simulation.h"
#include "aux.c"

//Macros to correctly access multidimensional arrays that has been flatten 
#define B(n, size_j, cord, i, j, pointer) pointer[(n * size_j * cord) + (i * size_j) + j]
#define S(size_i, size_j, cord, i, j, pointer) pointer[(size_i * size_j * cord) + (size_j * i) + j]

#define FILENAME_MAX_SIZE 256

/**
 * @struct Simulation
 * @brief Structure with the information for the generation of the N body simulation
 *
 * Structure declaration for the simulation, structured in the form
 * that the data is optimized to minimize cache misses in the CPU
 * and with all the data needed to use the GPU to do the calculations
 */
struct _Simulation
{   
    //Bodies variables
    double* masses;
    double* positions;
    double* velocity;
    int n;

    //Variables needed for internals of runge-kutta
    double* k1_position;
    double* k1_velocity;

    double* k2_position;
    double* k2_velocity;

    double* k3_position;
    double* k3_velocity;

    double* k4_position;
    double* k4_velocity;
    
    double* holder_position;
    double* holder_velocity;
    double* block_holder;
    
    //Cuda variables
    double* d_masses;
    double* d_position;
    double* d_k_velocity;
    dim3 threadBlockDims;
    dim3 gridDims;

} _Simulation;

//Internal helpers
int simulation_allocate_memory(Simulation* simulation);
int rk4(Simulation* simulation); 
int save_values_csv(Simulation* simulation, char* filename); 
int save_values_bin(Simulation* simulation, char* filename);
int calculate_kernel_size(Simulation* simulation);
int calculate_acceleration(Simulation* simulation, double*k_position, double* k_velocity);

//Cuda kernels
__global__ void calculate_acceleration_values_block_reduce(double* d_masses, double* d_position, double* d_block_holder, int n, double d_dt, unsigned int number_of_blocks_j);
__device__ void calculate_acceleration_values(double* d_masses, double* d_position, double* sdata, int n, double d_dt);
__device__ void full_block_reduction (double* d_block_holder, double* sdata, int n, unsigned int number_of_blocks_j);


#define cudaErrorCheck(ans) { gpuAssert((ans), __FILE__, __LINE__); }
/**
 * Function to check errors in CUDA. 
 * 
 * Extracted from StackOverflow 
 * @link https://stackoverflow.com/a/14038590
 */
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
    if (code != hipSuccess) 
    {
        fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
        if (abort) exit(code);
    }
}

Simulation* load_bodies(char* filepath)
/**
 * This funtion creates a new Simulation and fills it using the starting values from a file
 * @param filepath (char*):  a path to the file with the starting data, must be csv or bin file
 * @return simulation (Simulation*): a pointer to the new Simulation filled with the data in filepath
 */
{   
    //Allocate memory for the Simulation object itself
    Simulation* simulation = (Simulation*) malloc(sizeof(Simulation));
    if(simulation == NULL)
    {
        return NULL;
    }

    //Error checking
    if(filepath == NULL)
    {
        return NULL;
    }
    
    int extention_type = get_extention_type(filepath);
    if(extention_type == EXT_CSV)
    {   
        //Open file
        FILE* f = NULL;
        f = fopen(filepath, "r");
        if(f == NULL)
        {
            return NULL;
        }

        //Get the number of bodies by the number of lines minus the header
        simulation->n = count_lines_csv(f) - 1;
        if(simulation->n <= 0)
        {
            return NULL;
        }

        //Memory allocation for the arrays
        if(simulation_allocate_memory(simulation) != STATUS_OK)
        {
            return NULL;
        }

        //go back to the begining of file
        rewind(f);
        //For the number of bodies + header
        for(int i = 0; i < simulation->n + 1; i++)
        {     
            int j = i - 1;
            int joffset = j*3;
            
            //read header
            if(i == 0)
            {   
                //skip header line
                fscanf(f, "%*[^\n]\n");
                continue;
            }

            //Read bodies
            if(fscanf(f, "%lf,%lf,%lf,%lf,%lf,%lf,%lf,%*f\n", &simulation->positions[joffset], &simulation->positions[joffset+1], &simulation->positions[joffset+2], &simulation->masses[j], &simulation->velocity[joffset], &simulation->velocity[joffset+1], &(simulation->velocity[joffset+2])) == EOF)
            {
                printf("Error reading %s\n", filepath);
                return NULL;
            }
        }
        //close file
        fclose(f);
    }
    else if (extention_type == EXT_BIN)
    {
        //Read as binary
        FILE* f = fopen(filepath, "rb");
        if(f == NULL)
        {
            return NULL;
        }

        //Get file size
        fseek(f, 0, SEEK_END); 
        long size = ftell(f); 
        fseek(f, 0, SEEK_SET);

        //The number of bodies is the size of the file / size of each body
        simulation->n = size / (sizeof(double) * 8); 

        //Calculate kernel sizes
        if( calculate_kernel_size(simulation) == STATUS_ERROR )
        {
            free_simulation(simulation);
            return STATUS_ERROR;
        }

        //Memory allocation for the arrays
        if(simulation_allocate_memory(simulation) != STATUS_OK)
        {
            return NULL;
        }
        
        //Buffer for one body
        double buffer[8];
        //Read the whole file
        for (int i = 0; i < simulation->n; i++)
        {   
            int ioffset = i * 3;

            if(fread(buffer,sizeof(buffer),1,f) == 0)
                return NULL;

            simulation->positions[ioffset] = buffer[0];     //x
            simulation->positions[ioffset+1] = buffer[1];   //y
            simulation->positions[ioffset+2] = buffer[2];   //z
            simulation->masses[i] = buffer[3];              //mass
            simulation->velocity[ioffset] = buffer[4];      //vx
            simulation->velocity[ioffset+1] = buffer[5];    //vy
            simulation->velocity[ioffset+2] = buffer[6];    //vz

            //Buffer[7] is radius, currently useless for data, only useful for graphics
        }
        fclose(f);
        
    }
    else
    {
        return NULL;
    }
    
    //Copy masses to cuda memory
    hipError_t status = hipMemcpy( simulation->d_masses,  simulation->masses, simulation->n * sizeof(simulation->masses[0]),hipMemcpyHostToDevice);
    cudaErrorCheck(status);
    

    //Return simulation
    return simulation;
}

int simulation_allocate_memory(Simulation* simulation)
/**
 * Funtion that allocates all of the internal memory needed for the simulation
 * 
 * @param simulation (Simulation*): pointer to a fresh simulation in which all of the internal pointer still have to be allocated
 * @return status (int): STATUS_ERROR (0) in case of error STATUS_OK(1) in case everything when ok
*/
{   
    if(simulation == NULL || simulation->n <= 0)
        return STATUS_ERROR;

    simulation->masses = (double*) malloc (simulation->n * sizeof(simulation->masses[0]));
    simulation->positions = (double*) malloc (simulation->n * 3 * sizeof(simulation->positions[0]));
    simulation->velocity = (double*) malloc (simulation->n * 3 * sizeof(simulation->velocity[0]));

    simulation->k1_position = (double*) malloc (simulation->n * 3 * sizeof(simulation->k1_position[0]));
    simulation->k1_velocity = (double*) malloc (simulation->n * 3 * sizeof(simulation->k1_velocity[0]));

    simulation->k2_position = (double*) malloc (simulation->n * 3 * sizeof(simulation->k2_position[0]));
    simulation->k2_velocity = (double*) malloc (simulation->n * 3 * sizeof(simulation->k2_velocity[0]));

    simulation->k3_position = (double*) malloc (simulation->n * 3 * sizeof(simulation->k3_position[0]));
    simulation->k3_velocity = (double*) malloc (simulation->n * 3 * sizeof(simulation->k3_velocity[0]));

    simulation->k4_position = (double*) malloc (simulation->n * 3 * sizeof(simulation->k4_position[0]));
    simulation->k4_velocity = (double*) malloc (simulation->n * 3 * sizeof(simulation->k4_velocity[0]));

    simulation->holder_position = (double*) malloc (simulation->n * 3 * sizeof(simulation->holder_position[0]));
    simulation->holder_velocity = (double*) malloc (simulation->n * 3 * sizeof(simulation->holder_velocity[0]));

    simulation->block_holder = (double*) malloc (3 * simulation->n * ceil(simulation->n/32.0) * ceil(simulation->n/32.0) * sizeof(simulation->block_holder[0]));

    if(simulation->masses == NULL || simulation->block_holder == NULL
        || simulation->positions == NULL || simulation->velocity == NULL
        || simulation->k1_position == NULL || simulation->k1_velocity == NULL 
        || simulation->k2_position == NULL || simulation->k2_velocity == NULL 
        || simulation->k3_position == NULL || simulation->k3_velocity == NULL 
        || simulation->k4_position == NULL || simulation->k4_velocity == NULL 
        || simulation->holder_position == NULL || simulation->holder_velocity == NULL)
    {
        return STATUS_ERROR;
    }

    hipError_t status;

    //Cuda mallocs
    status = hipMalloc(&simulation->d_masses, simulation->n * sizeof(simulation->d_masses[0]));
    cudaErrorCheck(status);

    status = hipMalloc(&simulation->d_position, simulation->n * 3 * sizeof(simulation->d_position[0]));
    cudaErrorCheck(status);

    if(simulation->n <= 32)
        status = hipMalloc(&simulation->d_k_velocity, simulation->n * 3 * sizeof(simulation->d_k_velocity[0]));
    else
        status = hipMalloc(&simulation->d_k_velocity, simulation->n * simulation->gridDims.x * simulation->gridDims.y * 3 * sizeof(simulation->d_k_velocity[0]));
    cudaErrorCheck(status);

    return STATUS_OK;
}

void free_simulation(Simulation* simulation)
/**
 * This funtion frees all the memory used by the simulation
 * @param simulation (Simulation*):  a pointer to the simulation being set free
 */
{   
    //Frees all internal arrays
    free(simulation->masses);

    free(simulation->positions);
    free(simulation->velocity);

    free(simulation->k1_position);
    free(simulation->k1_velocity);

    free(simulation->k2_position);
    free(simulation->k2_velocity);

    free(simulation->k3_position);
    free(simulation->k3_velocity);

    free(simulation->k4_position);
    free(simulation->k4_velocity);

    free(simulation->holder_position);
    free(simulation->holder_velocity);
    free(simulation->block_holder);

    hipFree(simulation->d_masses);
    hipFree(simulation->d_position);
    hipFree(simulation->d_k_velocity);

    //Frees the simulation object itself
    free(simulation);
}

void print_simulation_values(Simulation* simulation)
/**
 * This function prints all of the values used in the simulation, used only for debugging purpuses
 * @param simulation (Simulation*):  a pointer to the simulation being printed
 */
{
    if(simulation == NULL)
    {
        return;
    }

    printf("Simulation Values...\nN: %d\n", simulation->n);
    printf("Masses: [");
    for(int i = 0; i < simulation->n; i++)
    {
        //Print last value
        if(i == simulation->n - 1)
            printf("%lf",simulation->masses[i]);
        //Print normal value
        else
            printf("%lf,",simulation->masses[i]);

    }
    printf("]\n");

    printf("Bodies: [");
    for(int i = 0; i < simulation->n; i++)
    {     
        int ioffset = i*3;
        //Print the last value from all bodies
        if(i == simulation->n - 1) 
            printf("%lf, %lf, %lf, %lf, %lf, %lf",simulation->positions[ioffset], simulation->positions[ioffset+1], simulation->positions[ioffset+2], simulation->velocity[ioffset], simulation->velocity[ioffset+1], simulation->velocity[ioffset+2]); 
        else
            printf("%lf, %lf, %lf, %lf, %lf, %lf, ",simulation->positions[ioffset], simulation->positions[ioffset+1], simulation->positions[ioffset+2], simulation->velocity[ioffset], simulation->velocity[ioffset+1], simulation->velocity[ioffset+2]); 
    }
    printf("]\n");
}

int save_values_csv(Simulation* simulation, char* filename)
/**
 * This funtion will print to the file f the current positions of all the bodies in the simulation as a csv
 * @param simulation (Simulation*):  a pointer to the simulation being stored
 * @param file (char*) the filepath in which the data is going to be stored as csv
 * @return status (int): STATUS_ERROR (0) in case of error STATUS_OK(1) in case everything when ok
 */
{   
    //Error checking
    if(simulation == NULL || filename == NULL)
        return STATUS_ERROR;

    //Open file
    FILE* f = fopen(filename, "w");
    if(f == NULL)
        return STATUS_ERROR;

    //For all n bodies
    for(int i = 0; i < simulation->n; i++)
    {      
        //Print body as csv x,y,z
        int ioffset = i*3;
        if(fprintf(f, "%lf,%lf,%lf\n", simulation->positions[ioffset], simulation->positions[ioffset+1], simulation->positions[ioffset+2]) < 0)
            return STATUS_ERROR;
    }

    fclose(f);
    return STATUS_OK;
}

int save_values_bin(Simulation* simulation, char* filename)
/**
 * This funtion will print to the file f the current positions of all the bodies in the simulation as a bin
 * @param simulation (Simulation*):  a pointer to the simulation being stored
 * @param file (char*) the filepath in which the data is going to be stored as bin
 * @return status (int): STATUS_ERROR (0) in case of error STATUS_OK(1) in case everything when ok
 */
{   
    //Error checking
    if(simulation == NULL || filename == NULL)
        return STATUS_ERROR;

    //Open file
    FILE* f = fopen(filename, "wb");
    if(f == NULL)
        return STATUS_ERROR;

    double buffer[3];

    //For all n bodies
    for(int i = 0; i < simulation->n; i++)
    {      
        int ioffset = i*3;

        buffer[0] = simulation->positions[ioffset];
        buffer[1] = simulation->positions[ioffset+1];
        buffer[2] =  simulation->positions[ioffset+2];
        
        //write body as bin x,y,z
        if(fwrite(buffer, sizeof(buffer), 1, f) == 0)
            return STATUS_ERROR;
    }

    fclose(f);
    return STATUS_OK;
}

double run_simulation(Simulation* simulation, double T)
/**
 * Funtion that will run the simulation for T internal seconds (this means that the ending positions of the bodies will be in time T)
 *
 * This funtion will calculate the positions of the bodies every in timesteps of 'dt'using the runge-kutta method
 * and store them in data/ as csv files every 'speed' seconds
 * 
 * @param simulation (Simulation*) pointer to the simulation object with the initial values       
 * @param T (double): Internal ending time of the simulation
 * 
 * @return t (double): Real time that the simulation was running, STATUS_ERROR in case something went wrong
**/
{   
    //Calculate the number of steps we will have to take to get to T
    long int steps = T / dt;
    //Calculate the number of timesteps we must do before saving the data
    long int save_step = speed / dt;
    //Internal variables to keep track of csv files written
    long int file_number = 1;

    char filename[FILENAME_MAX_SIZE];

    //Internal variables to measure time 
    struct timeval t_start, t_end;
    gettimeofday ( &t_start, NULL );

    printf("Simulating with CUDA\n");

    //Run simulation
    for(long int step = 1; step <= steps; step++)
    {
        //Integrate next step using runge-kutta
        if(rk4(simulation) == STATUS_ERROR)
            return STATUS_ERROR;
        
        //Save data if we must
        if(step % save_step == 0)
        {   
            if(snprintf(filename, FILENAME_MAX, "../Graphics/data/%ld.bin", file_number) < 0)
                return STATUS_ERROR;

            if(save_values_bin(simulation, filename) == STATUS_ERROR)
                return STATUS_ERROR;
            file_number++;
        }

        //Print fancy progress 
        printf("\rIntegrating: step = %ld / %ld", step, steps);
	    fflush(stdout);
    }
    
    //Calculate how long the simulation took
    gettimeofday ( &t_end, NULL );
    printf("\nSimulation completed in %lf seconds\n",  WALLTIME(t_end) - WALLTIME(t_start));
    return WALLTIME(t_end) - WALLTIME(t_start);
}

int rk4(Simulation* simulation)
/**
 * This funtion will calculate the next values of the simulation using the runge-kutta method
 * 
 * @param simulation (Simulation*): a pointer to the simulation
 * @return status (int): STATUS_ERROR (0) in case of error STATUS_OK(1) in case everything when ok
 **/
{   
    //Correctly set up holder
    for(int i = 0; i < simulation->n * 3; i++)
    {
        simulation->holder_position[i] = simulation->positions[i];
        simulation->holder_velocity[i] = simulation->velocity[i];
    }
    
    //Calculate k1
    if(calculate_acceleration(simulation, simulation->k1_position, simulation->k1_velocity) == STATUS_ERROR)
        return STATUS_ERROR;

    //Calculate simulation.bodies+0.5*k1 to be able to calculate k2
    for(int i = 0; i < simulation->n * 3; i++)
    {   
        simulation->holder_position[i] = simulation->positions[i] + simulation->k1_position[i] * 0.5;
        simulation->holder_velocity[i] = simulation->velocity[i] + simulation->k1_velocity[i] * 0.5;
    }

    //Calculate k2
    if(calculate_acceleration(simulation, simulation->k2_position, simulation->k2_velocity) == STATUS_ERROR)
        return STATUS_ERROR;

    //Calculate simulation.bodies+0.5*k2 to be able to calculate k3
    for(int i = 0; i < simulation->n * 3; i++)
    {
        simulation->holder_position[i] = simulation->positions[i] + simulation->k2_position[i] * 0.5;
        simulation->holder_velocity[i] = simulation->velocity[i] + simulation->k2_velocity[i] * 0.5;
    }

    //Calculate k3
    if(calculate_acceleration(simulation, simulation->k3_position, simulation->k3_velocity) == STATUS_ERROR)
        return STATUS_ERROR;

    //Calculate simulation.bodies+*k3 to be able to calculate k3
    for(int i = 0; i < simulation->n * 3; i++)
    {
        simulation->holder_position[i] = simulation->positions[i] + simulation->k3_position[i];
        simulation->holder_velocity[i] = simulation->velocity[i] + simulation->k3_velocity[i];
    }

    //Calculate k4
    if(calculate_acceleration(simulation, simulation->k4_position, simulation->k4_velocity) == STATUS_ERROR)
        return STATUS_ERROR;

    //Update simulation value to simulation.bodies + ((k1 + 2*k2 + 2*k3 + k4) / 6.0)
    for(int i = 0; i < simulation->n * 3; i++)
    {
        simulation->positions[i] = simulation->positions[i] + ((simulation->k1_position[i] + 2.0*simulation->k2_position[i] + 2.0*simulation->k3_position[i] + simulation->k4_position[i]) / 6.0);
        simulation->velocity[i] = simulation->velocity[i] + ((simulation->k1_velocity[i] + 2.0*simulation->k2_velocity[i] + 2.0*simulation->k3_velocity[i] + simulation->k4_velocity[i]) / 6.0);
    }

    return STATUS_OK;
}


int calculate_acceleration(Simulation* simulation, double*k_position, double* k_velocity)
/**
 * Funtion to calculate the velocity and acceleration of the bodies using the current positions and velocities. It uses a cuda kernel to calculate the values
 * @param simulation(Simulation*): a pointer to the simulation object we are simulating
 * @param k_position (double*): Array to store resulting positions of the N bodies. They are stored as follows x1,y1,z1,x2,y2,z2....xn,yn,zn
 * @param k_velocity (double*): Array to store the resulting velocities of the N bodies. They are stored as follows vx1,vy1,vz1,vx2,vy2,vz2....vxn,vyn,vzn
 * 
 * @return status (int): STATUS_ERROR (0) in case of error STATUS_OK(1) in case everything when ok
**/
{   
    //Error checking
    if(simulation == NULL || k_position == NULL|| k_velocity == NULL)
        return STATUS_ERROR;

    //Init values of k
    for(int i = 0; i < simulation->n; i++)
    {   
        int ioffset = i * 3;
        k_position[ioffset] = dt * simulation->holder_velocity[ioffset];
        k_position[ioffset+1] = dt * simulation->holder_velocity[ioffset+1];
        k_position[ioffset+2] = dt * simulation->holder_velocity[ioffset+2];
        k_velocity[ioffset] = 0.0;
        k_velocity[ioffset+1] = 0.0;
        k_velocity[ioffset+2] = 0.0;
    }

    //Set up memory for cuda
    hipMemcpy( simulation->d_position,  simulation->holder_position, simulation->n * 3 * sizeof(simulation->holder_position[0]),hipMemcpyHostToDevice);
    hipMemset( simulation->d_k_velocity, 0.0, 3 * simulation->n  * simulation->gridDims.y * sizeof(simulation->d_k_velocity[0]));
    
    //Call cuda
    calculate_acceleration_values_block_reduce<<<simulation->gridDims, simulation->threadBlockDims, 3 * simulation->threadBlockDims.x * simulation->threadBlockDims.y * sizeof(double)>>>(simulation->d_masses, simulation->d_position, simulation->d_k_velocity ,simulation->n, dt, simulation->gridDims.y);
    hipError_t status = hipGetLastError();
    cudaErrorCheck(status);

    //Pass results to cpu
    hipMemcpy( simulation->block_holder, simulation->d_k_velocity, 3 * simulation->n * simulation->gridDims.y * sizeof(simulation->block_holder[0]), hipMemcpyDeviceToHost);
    
    //Aggregate results
    for(int i = 0; i < simulation->n; i++)
    {
        int ioffset = i * 3;
        for(int j = 0; j < simulation->gridDims.y; j++)
        {
            k_velocity[ioffset] += B(simulation->n,simulation->gridDims.y, 0, i, j, simulation->block_holder);
            k_velocity[ioffset + 1] += B(simulation->n,simulation->gridDims.y, 1, i, j, simulation->block_holder);
            k_velocity[ioffset + 2] += B(simulation->n,simulation->gridDims.y, 2, i, j, simulation->block_holder);
        } 
    }
    return STATUS_OK;
}

__device__ void calculate_acceleration_values(double* d_masses, double* d_position, double* sdata, int n, double d_dt)
/**
 * Cuda kernel that calculates the acceleration values that each body suffers from every other body
 * @param d_masses (double*): A cuda array of size n with the masses of each body
 * @param d_position (double*): A cuda array of size 3n with the current position of all of the bodies stored in the following matter x1,y1,z1,x2,y2,z2...xn,yn,zn
 * @param sdata (double*): A shared array in which to store all of the acceleration values
 * @param n (int): the number of bodies
 * @param d_dt (double): the timestep increment
 */
{
    //Get position in the block
    int x = threadIdx.x;
    int y = threadIdx.y;

    //Get universal position
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    int j = threadIdx.y + blockIdx.y * blockDim.y;

    int ioffset = i * 3;
    int joffset = j * 3;
    
    
    //Calculate pull of one body by other body
    if(i < n && j < n && i != j)
    {   
        double dx = d_position[joffset] - d_position[ioffset]; //rx body 2 - rx body 1
        double dy = d_position[joffset+1] - d_position[ioffset+1]; //ry body 2 - ry body 1
        double dz = d_position[joffset+2] - d_position[ioffset+2]; //rz body 2 - rz body 1
        
        double r = dx * dx + dy * dy + dz * dz + softening * softening; //distance magnitud with some softening
        double h = ((G * d_masses[j]) / (pow(r, 1.5))); //Acceleration formula

        S(blockDim.x, blockDim.y, 0, x, y, sdata) = d_dt * h * dx; //Acceleration formula for x
        S(blockDim.x, blockDim.y, 1, x, y, sdata) = d_dt * h * dy; //Acceleration formula for y
        S(blockDim.x, blockDim.y, 2, x, y, sdata) = d_dt * h * dz; //Acceleration formula for z
    }
    //Fill with 0 the remaining values in the array with 0
    else
    {  
        S(blockDim.x, blockDim.y, 0, x, y, sdata) = 0.0;    //x
        S(blockDim.x, blockDim.y, 1, x, y, sdata) = 0.0;    //y
        S(blockDim.x, blockDim.y, 2, x, y, sdata) = 0.0;    //z
    }
    
}
__global__ void calculate_acceleration_values_block_reduce(double* d_masses, double* d_position, double* d_block_holder, int n, double d_dt, unsigned int number_of_blocks_j)
/**
 * A cuda kernel that calculate the 3n**2 acceleration values and reduce them to an array of size of 3n * gridDim.y. 
 * @param d_masses (double*): A cuda array of size n with the masses of each body
 * @param d_position (double*): A cuda array of size 3n with the current position of all of the bodies stored in the following matter x1,y1,z1,x2,y2,z2...xn,yn,zn
 * @param d_block_holder (double*): An array where the block-reduced results will be stored. It must be of size 3n * gridDim.y
 * @param n (int): the number of bodies
 * @param d_dt (double): the timestep increment
 * @param number_of_blocks_j (unsigned int): The number of blocks in the j dimension. Used for the block reduction
 */
{   
    //Array where all values of this block will be stored
    extern __shared__ double sdata[];

    //Calculate aceleration values
    calculate_acceleration_values(d_masses, d_position, sdata, n, d_dt);
        
    //Reduce all values of this block
    full_block_reduction(d_block_holder, sdata, n, number_of_blocks_j);
}

int calculate_kernel_size(Simulation* simulation)
/**
 * A simple funtion to calculate the most efficient kernel sizes for this simulation depending of the size of N
 * @param simulation (Simulation*): a pointer to the simulation
 */
{
    if(simulation == NULL)
        return STATUS_ERROR;
    
    unsigned int x = 32;
    unsigned int y = 32;

    for(; y <= 1024; x/=2, y*=2)
    {
        if(simulation->n <= y)
        {
            simulation->threadBlockDims = {x, y, 1} ; //1024 threads per block
            simulation->gridDims = { (unsigned int) ceil(simulation->n/(double) x), (unsigned int) ceil( simulation->n/(double) y), 1 }; 
            return STATUS_OK;
        }
    }

    x = 1;
    y = 1024;

    simulation->threadBlockDims = {x, y, 1} ; //1024 threads per block
    simulation->gridDims = { (unsigned int) ceil(simulation->n/(double) x), (unsigned int) ceil( simulation->n/(double) y), 1 }; 

    return STATUS_OK;
}

__device__ void full_block_reduction (double* d_block_holder, double* sdata, int n, unsigned int number_of_blocks_j)
/**
 * A cuda kernel that reduces the acceleration values of this block to one for every body in this block
 * @param d_block_holder (double*): An array where the block-reduced results will be stored. It must be of size 3n * gridDim.y
 * @param sdata (double*): A shared array in which all of the aceleration values of this block are stored
 * @param n (int): the number of bodies
 * @param number_of_blocks_j (unsigned int): The number of blocks in the j dimension.
 */
{
    //Get position in the block
    int x = threadIdx.x;
    int y = threadIdx.y;

    //Get universal position
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    
    __syncthreads();

    // do reduction in shared mem
    for (unsigned int s=blockDim.y/2; s>0; s>>=1) 
    {
        if (y < s) 
        {
            S(blockDim.x, blockDim.y, 0, x, y, sdata) += S(blockDim.x, blockDim.y, 0, x, y + s, sdata);
            S(blockDim.x, blockDim.y, 1, x, y, sdata) += S(blockDim.x, blockDim.y, 1, x, y + s, sdata);
            S(blockDim.x, blockDim.y, 2, x, y, sdata) += S(blockDim.x, blockDim.y, 2, x, y + s, sdata);
        }
        __syncthreads();
    }
    // write result for this block to global mem
    if (y == 0)
    {
        B(n, number_of_blocks_j, 0, i, blockIdx.y, d_block_holder) += S(blockDim.x, blockDim.y, 0, x, 0, sdata);
        B(n, number_of_blocks_j, 1, i, blockIdx.y, d_block_holder) += S(blockDim.x, blockDim.y, 1, x, 0, sdata);
        B(n, number_of_blocks_j, 2, i, blockIdx.y, d_block_holder) += S(blockDim.x, blockDim.y, 2, x, 0, sdata);
    } 
}

