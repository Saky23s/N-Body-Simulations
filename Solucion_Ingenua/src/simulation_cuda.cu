#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include<unistd.h>
#include <time.h>
#include <string.h>
#include <sys/time.h>
#include "../inc/simulation.h"
#include "aux.c"

#define B(n, block_size, cord, i, block_i, pointer) pointer[(n * block_size  * cord) + (i * block_size) + block_i]
#define S(size_i, size_j, cord, i, j, pointer) pointer[(size_i * size_j * cord) + (size_i * i) + j]

struct _Simulation
{
    double* masses;
    double* positions;
    double* velocity;
    int n;

    //Variables needed for internals
    double* k1_position;
    double* k1_velocity;

    double* k2_position;
    double* k2_velocity;

    double* k3_position;
    double* k3_velocity;

    double* k4_position;
    double* k4_velocity;
    
    double* holder_position;
    double* holder_velocity;
    double* block_holder;
    
    //Cuda variables
    double* d_masses;
    double* d_acceleration;
    double* d_holder_position;
    double* d_k_velocity;
} _Simulation;



//Internal helpers
int simulation_allocate_memory(Simulation* simulation);
int rk4(Simulation* simulation); 
int save_values_csv(Simulation* simulation, char* filename); 
int save_values_bin(Simulation* simulation, char* filename);
int calculate_acceleration(Simulation* simulation, double*k_position,double*k_velocity);

__global__ void calculate_acceleration_values_block_reduce(double* d_masses, double* d_holder_position, double* d_aceleration, double* d_block_holder, int n, double d_dt, unsigned int block_n);

#define cudaErrorCheck(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
    if (code != hipSuccess) 
    {
        fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
        if (abort) exit(code);
    }
}

Simulation* load_bodies(char* filepath)
/**
 * This funtion creates a new Simulation and fills it using the starting values from a file
 * @param filepath (char*):  a path to the file with the starting data, must be csv or bin file
 * @return simulation (Simulation*): a pointer to the new Simulation filled with the data in filepath
 */
{   
    //Allocate memory for the Simulation object itself
    Simulation* simulation = (Simulation*) malloc(sizeof(Simulation));
    if(simulation == NULL)
    {
        return NULL;
    }

    //Error checking
    if(filepath == NULL)
    {
        return NULL;
    }
    
    
    int extention_type = get_extention_type(filepath);
    if(extention_type == EXT_CSV)
    {   
        //Open file
        FILE* f = NULL;
        f = fopen(filepath, "r");
        if(f == NULL)
        {
            return NULL;
        }

        //Get the number of bodies by the number of lines minus the header
        simulation->n = count_lines_csv(f) - 1;
        if(simulation->n <= 0)
        {
            return NULL;
        }

        //Memory allocation for the arrays
        if(simulation_allocate_memory(simulation) != STATUS_OK)
        {
            return NULL;
        }

        //go back to the begining of file
        rewind(f);
        //For the number of bodies + header
        for(int i = 0; i < simulation->n + 1; i++)
        {     
            int j = i - 1;
            int joffset = j*3;
            //read header
            if(i == 0)
            {   
                //skip header line
                fscanf(f, "%*[^\n]\n");
                continue;
            }

            fscanf(f, "%lf,%lf,%lf,%lf,%lf,%lf,%lf,%*f\n", &simulation->positions[joffset], &simulation->positions[joffset+1], &simulation->positions[joffset+2], &simulation->masses[j], &simulation->velocity[joffset], &simulation->velocity[joffset+1], &(simulation->velocity[joffset+2]));

        }
        //close file
        fclose(f);
    }
    else if (extention_type == EXT_BIN)
    {
        //Read as binary
        FILE* f = fopen(filepath, "rb");
        if(f == NULL)
        {
            return NULL;
        }

        //Get file size
        fseek(f, 0, SEEK_END); 
        long size = ftell(f); 
        fseek(f, 0, SEEK_SET);

        //The number of bodies is the size of the file / size of each body
        simulation->n = size / (sizeof(double) * 8); 

        //Memory allocation for the arrays
        if(simulation_allocate_memory(simulation) != STATUS_OK)
        {
            return NULL;
        }
        
        //Buffer for one body
        double buffer[8];
        //Read the whole file
        for (int i = 0; i < simulation->n; i++)
        {   
            int ioffset = i * 3;
            fread(buffer,sizeof(buffer),1,f);
            simulation->positions[ioffset] = buffer[0];     //x
            simulation->positions[ioffset+1] = buffer[1];   //y
            simulation->positions[ioffset+2] = buffer[2];   //z
            simulation->masses[i] = buffer[3];              //mass
            simulation->velocity[ioffset] = buffer[4];      //vx
            simulation->velocity[ioffset+1] = buffer[5];    //vy
            simulation->velocity[ioffset+2] = buffer[6];    //vz

            //Buffer[7] is radius, currently useless for data, only useful for graphics
        }
        fclose(f);
        
    }
    else
    {
        return NULL;
    }
    
    //Copy masses to cuda memory
    hipMemcpy( simulation->d_masses,  simulation->masses, simulation->n * sizeof(simulation->masses[0]),hipMemcpyHostToDevice);
    
    //Return simulation
    return simulation;
}

int simulation_allocate_memory(Simulation* simulation)
/**
 * Funtion that allocates all of the internal arrays of the simulation
 * 
 * @param simulation (Simulation*): pointer to a fresh simulation in which all of the internal pointer still have to be allocated
 * @return status (int): STATUS_ERROR (0) in case of error STATUS_OK(1) in case everything when ok
*/
{   
    if(simulation == NULL || simulation->n <= 0)
        return STATUS_ERROR;

    simulation->masses = (double*) malloc (simulation->n * sizeof(simulation->masses[0]));
    simulation->positions = (double*) malloc (simulation->n * 3 * sizeof(simulation->positions[0]));
    simulation->velocity = (double*) malloc (simulation->n * 3 * sizeof(simulation->velocity[0]));

    simulation->k1_position = (double*) malloc (simulation->n * 3 * sizeof(simulation->k1_position[0]));
    simulation->k1_velocity = (double*) malloc (simulation->n * 3 * sizeof(simulation->k1_velocity[0]));

    simulation->k2_position = (double*) malloc (simulation->n * 3 * sizeof(simulation->k2_position[0]));
    simulation->k2_velocity = (double*) malloc (simulation->n * 3 * sizeof(simulation->k2_velocity[0]));

    simulation->k3_position = (double*) malloc (simulation->n * 3 * sizeof(simulation->k3_position[0]));
    simulation->k3_velocity = (double*) malloc (simulation->n * 3 * sizeof(simulation->k3_velocity[0]));

    simulation->k4_position = (double*) malloc (simulation->n * 3 * sizeof(simulation->k4_position[0]));
    simulation->k4_velocity = (double*) malloc (simulation->n * 3 * sizeof(simulation->k4_velocity[0]));

    simulation->holder_position = (double*) malloc (simulation->n * 3 * sizeof(simulation->holder_position[0]));
    simulation->holder_velocity = (double*) malloc (simulation->n * 3 * sizeof(simulation->holder_velocity[0]));

    simulation->block_holder = (double*) malloc (3 * simulation->n * ceil(simulation->n/32.0) * ceil(simulation->n/32.0) * sizeof(simulation->block_holder[0]));

    if(simulation->masses == NULL || simulation->block_holder == NULL
        || simulation->positions == NULL || simulation->velocity == NULL
        || simulation->k1_position == NULL || simulation->k1_velocity == NULL 
        || simulation->k2_position == NULL || simulation->k2_velocity == NULL 
        || simulation->k3_position == NULL || simulation->k3_velocity == NULL 
        || simulation->k4_position == NULL || simulation->k4_velocity == NULL 
        || simulation->holder_position == NULL || simulation->holder_velocity == NULL)
    {
        return STATUS_ERROR;
    }

    hipError_t status;

    //Cuda mallocs
    status = hipMalloc(&simulation->d_masses, simulation->n * sizeof(simulation->d_masses[0]));
    cudaErrorCheck(status);

    status = hipMalloc(&simulation->d_acceleration, 3 * simulation->n * simulation->n * sizeof(simulation->d_acceleration[0]));
    cudaErrorCheck(status);

    status = hipMalloc(&simulation->d_holder_position, simulation->n * 3 * sizeof(simulation->d_holder_position[0]));
    cudaErrorCheck(status);

    if(simulation->n <= 32)
        status = hipMalloc(&simulation->d_k_velocity, simulation->n * 3 * sizeof(simulation->d_k_velocity[0]));
    else
        status = hipMalloc(&simulation->d_k_velocity, simulation->n * ceil(simulation->n / 32.0) * ceil(simulation->n / 32.0) * 3 * sizeof(simulation->d_k_velocity[0]));
    cudaErrorCheck(status);

    return STATUS_OK;
}

void free_simulation(Simulation* simulation)
/**
 * This funtion frees all the memory used by the simulation
 * @param simulation (Simulation*):  a pointer to the simulation being set free
 */
{   
    //Frees all internal arrays
    free(simulation->masses);

    free(simulation->positions);
    free(simulation->velocity);

    free(simulation->k1_position);
    free(simulation->k1_velocity);

    free(simulation->k2_position);
    free(simulation->k2_velocity);

    free(simulation->k3_position);
    free(simulation->k3_velocity);

    free(simulation->k4_position);
    free(simulation->k4_velocity);

    free(simulation->holder_position);
    free(simulation->holder_velocity);

    if(simulation->block_holder)
        free(simulation->block_holder);

    hipFree(simulation->d_acceleration);
    hipFree(simulation->d_masses);
    hipFree(simulation->d_holder_position);
    hipFree(simulation->d_k_velocity);

    //Frees the simulation object itself
    free(simulation);
}

void print_simulation_values(Simulation* simulation)
/**
 * This funtion prints all of the valkues used by the simulation
 * @param simulation (Simulation*):  a pointer to the simulation being printed
 */
{
    if(simulation == NULL)
    {
        return;
    }

    printf("Simulation Values...\nN: %d\n", simulation->n);
    printf("Masses: [");
    for(int i = 0; i < simulation->n; i++)
    {
        //Print last value
        if(i == simulation->n - 1)
            printf("%lf",simulation->masses[i]);
        //Print normal value
        else
            printf("%lf,",simulation->masses[i]);

    }
    printf("]\n");

    printf("Bodies: [");
    for(int i = 0; i < simulation->n; i++)
        {     
            int ioffset = i*3;
            //Print the last value from all bodies
            if(i == simulation->n - 1) 
                printf("%lf, %lf, %lf, %lf, %lf, %lf",simulation->positions[ioffset], simulation->positions[ioffset+1], simulation->positions[ioffset+2], simulation->velocity[ioffset], simulation->velocity[ioffset+1], simulation->velocity[ioffset+2]); 
            else
                printf("%lf, %lf, %lf, %lf, %lf, %lf, ",simulation->positions[ioffset], simulation->positions[ioffset+1], simulation->positions[ioffset+2], simulation->velocity[ioffset], simulation->velocity[ioffset+1], simulation->velocity[ioffset+2]); 
        }
    printf("]\n");
}

int save_values_csv(Simulation* simulation, char* filename)
/**
 * This funtion will print to the file f the current positions of all the bodies in the simulation as a csv
 * @param simulation (Simulation*):  a pointer to the simulation being stored
 * @param file (char*) the filepath in which the data is going to be stored as csv
 * @return status (int): STATUS_ERROR (0) in case of error STATUS_OK(1) in case everything when ok
 */
{   
    //Error checking
    if(simulation == NULL || filename == NULL)
        return STATUS_ERROR;

    //Open file
    FILE* f = fopen(filename, "w");
    if(f == NULL)
        return STATUS_ERROR;

    //For all n bodies
    for(int i = 0; i < simulation->n; i++)
    {      
        //Print body as csv x,y,z
        int ioffset = i*3;
        fprintf(f, "%lf,%lf,%lf\n", simulation->positions[ioffset], simulation->positions[ioffset+1], simulation->positions[ioffset+2]);
    }

    fclose(f);
    return STATUS_OK;
}

int save_values_bin(Simulation* simulation, char* filename)
/**
 * This funtion will print to the file f the current positions of all the bodies in the simulation as a bin
 * @param simulation (Simulation*):  a pointer to the simulation being stored
 * @param file (char*) the filepath in which the data is going to be stored as bin
 * @return status (int): STATUS_ERROR (0) in case of error STATUS_OK(1) in case everything when ok
 */
{   
    //Error checking
    if(simulation == NULL || filename == NULL)
        return STATUS_ERROR;

    //Open file
    FILE* f = fopen(filename, "wb");
    if(f == NULL)
        return STATUS_ERROR;

    double buffer[3];

    //For all n bodies
    for(int i = 0; i < simulation->n; i++)
    {      
        int ioffset = i*3;

        buffer[0] = simulation->positions[ioffset];
        buffer[1] = simulation->positions[ioffset+1];
        buffer[2] =  simulation->positions[ioffset+2];
        
        //write body as bin x,y,z
        fwrite(buffer, sizeof(buffer), 1, f);
    }

    fclose(f);
    return STATUS_OK;
}

double run_simulation(Simulation* simulation, double T)
/**
 * Funtion that will run the simulation for T internal seconds (this means that the ending positions of the bodies will be in time T)
 *
 * This funtion will calculate the positions of the bodies every in timesteps of 'dt'using the runge-kutta method
 * and store them in data/ as csv files every 'speed' seconds
 * 
 * @param simulation (Simulation*) pointer to the simulation object with the initial values       
 * @param T (double): Internal ending time of the simulation
 * 
 * @return t (double): Real time that the simulation was running, STATUS_ERROR in case something went wrong
**/
{   
    //Calculate the number of steps we will have to take to get to T
    long int steps = T / dt;
    //Calculate the number of timesteps we must do before saving the data
    long int save_step = speed / dt;
    //Internal variables to keep track of csv files written
    long int file_number = 1;

    char filename[256];

    //Internal variables to measure time 
    struct timeval t_start, t_end;
    gettimeofday ( &t_start, NULL );

    printf("Simulating with CUDA\n");

    //Run simulation
    for(long int step = 1; step <= steps; step++)
    {
        //Integrate next step using runge-kutta
        if(rk4(simulation) == STATUS_ERROR)
            return STATUS_ERROR;
        
        //Save data if we must
        if(step % save_step == 0)
        {   
            sprintf(filename, "../Graphics/data/%ld.bin", file_number);
            if(save_values_bin(simulation, filename) == STATUS_ERROR)
                return STATUS_ERROR;
            file_number++;
        }

        //Print fancy progress 
        printf("\rIntegrating: step = %ld / %ld", step, steps);
	    fflush(stdout);
    }
    
    //Calculate how long the simulation took
    gettimeofday ( &t_end, NULL );
    printf("\nSimulation completed in %lf seconds\n",  WALLTIME(t_end) - WALLTIME(t_start));
    return WALLTIME(t_end) - WALLTIME(t_start);
}

int rk4(Simulation* simulation)
/**
 * This funtion will calculate the next values of the simulation using the runge-kutta method
 * 
 * @param simulation (Simulation*): a pointer to the simulation
 * @return status (int): STATUS_ERROR (0) in case of error STATUS_OK(1) in case everything when ok
 **/
{   
    //Correctly set up holder
    for(int i = 0; i < simulation->n * 3; i++)
    {
        simulation->holder_position[i] = simulation->positions[i];
        simulation->holder_velocity[i] = simulation->velocity[i];

    }
    
    //Calculate k1
    if(calculate_acceleration(simulation, simulation->k1_position, simulation->k1_velocity) == STATUS_ERROR)
        return STATUS_ERROR;

    //Calculate simulation.bodies+0.5*k1 to be able to calculate k2
    for(int i = 0; i < simulation->n * 3; i++)
    {   
        simulation->holder_position[i] = simulation->positions[i] + simulation->k1_position[i] * 0.5;
        simulation->holder_velocity[i] = simulation->velocity[i] + simulation->k1_velocity[i] * 0.5;
    }

    //Calculate k2
    if(calculate_acceleration(simulation, simulation->k2_position, simulation->k2_velocity) == STATUS_ERROR)
        return STATUS_ERROR;

    //Calculate simulation.bodies+0.5*k2 to be able to calculate k3
    for(int i = 0; i < simulation->n * 3; i++)
    {
        simulation->holder_position[i] = simulation->positions[i] + simulation->k2_position[i] * 0.5;
        simulation->holder_velocity[i] = simulation->velocity[i] + simulation->k2_velocity[i] * 0.5;
    }

    //Calculate k3
    if(calculate_acceleration(simulation, simulation->k3_position, simulation->k3_velocity) == STATUS_ERROR)
        return STATUS_ERROR;

    //Calculate simulation.bodies+*k3 to be able to calculate k3
    for(int i = 0; i < simulation->n * 3; i++)
    {
        simulation->holder_position[i] = simulation->positions[i] + simulation->k3_position[i];
        simulation->holder_velocity[i] = simulation->velocity[i] + simulation->k3_velocity[i];
    }

    //Calculate k4
    if(calculate_acceleration(simulation, simulation->k4_position, simulation->k4_velocity) == STATUS_ERROR)
        return STATUS_ERROR;

    //Update simulation value to simulation.bodies + ((k1 + 2*k2 + 2*k3 + k4) / 6.0)
    for(int i = 0; i < simulation->n * 3; i++)
    {
        simulation->positions[i] = simulation->positions[i] + ((simulation->k1_position[i] + 2.0*simulation->k2_position[i] + 2.0*simulation->k3_position[i] + simulation->k4_position[i]) / 6.0);
        simulation->velocity[i] = simulation->velocity[i] + ((simulation->k1_velocity[i] + 2.0*simulation->k2_velocity[i] + 2.0*simulation->k3_velocity[i] + simulation->k4_velocity[i]) / 6.0);
    }

    return STATUS_OK;
}


int calculate_acceleration(Simulation* simulation, double*k_position, double* k_velocity)
/**
 * Funtion to calculate the velocity and acceleration of the bodies using the current positions and velocities
 * @param simulation(Simulation*): a pointer to the simulation object we are simulation, in the holder variable the information must be stored as an array of values order as x1,y1,z1,vx1,vz1,vz1,x2,y2,z2,vx2,vz2,vz2...xn,yn,zn,vxn,vzn,vzn
 * @return status (int): STATUS_ERROR (0) in case of error STATUS_OK(1) in case everything when ok
**/
{   
    //Error checking
    if(simulation == NULL || k_position == NULL|| k_velocity == NULL)
        return STATUS_ERROR;

    //Init values of k
    for(int i = 0; i < simulation->n; i++)
    {   
        int ioffset = i * 3;
        k_position[ioffset] = dt * simulation->holder_velocity[ioffset];
        k_position[ioffset+1] = dt * simulation->holder_velocity[ioffset+1];
        k_position[ioffset+2] = dt * simulation->holder_velocity[ioffset+2];
        k_velocity[ioffset] = 0.0;
        k_velocity[ioffset+1] = 0.0;
        k_velocity[ioffset+2] = 0.0;
    }

    //Call cuda
    dim3 threadBlockDims;
    dim3 gridDims;
    unsigned int block_n;
    
    if(simulation->n <= 32)
    {
        threadBlockDims= {(unsigned int) simulation->n , (unsigned int) simulation->n , 1 } ; //1024 threads per block
        gridDims = { 1, 1, 1 } ; 
        block_n = 1;
    }
    else
    {
        threadBlockDims = {32 , 32 , 1 } ; //1024 threads per block
        block_n = (unsigned int) ceil( simulation->n/32.0);
        gridDims = { block_n, block_n, 1 };
    }

    hipMemcpy( simulation->d_holder_position,  simulation->holder_position, simulation->n * 3 * sizeof(simulation->holder_position[0]),hipMemcpyHostToDevice);
    hipMemset( simulation->d_k_velocity, 0.0, 3 * simulation->n  * block_n * sizeof(simulation->d_k_velocity[0]));
    
    calculate_acceleration_values_block_reduce<<<gridDims, threadBlockDims, 3 * threadBlockDims.x * threadBlockDims.y * sizeof(double)>>>(simulation->d_masses, simulation->d_holder_position, simulation->d_acceleration, simulation->d_k_velocity ,simulation->n, dt, block_n);
    hipError_t status = hipGetLastError();
    cudaErrorCheck(status);

    hipMemcpy( simulation->block_holder, simulation->d_k_velocity, 3 * simulation->n * block_n* sizeof(simulation->block_holder[0]), hipMemcpyDeviceToHost);
    
    for(int i = 0; i < simulation->n; i++)
    {
        int ioffset = i * 3;
        for(int block_i = 0; block_i < block_n; block_i++)
        {
            
            k_velocity[ioffset] += B(simulation->n,block_n, 0, i, block_i, simulation->block_holder);
            k_velocity[ioffset + 1] += B(simulation->n,block_n, 1, i, block_i, simulation->block_holder);
            k_velocity[ioffset + 2] += B(simulation->n,block_n, 2, i, block_i, simulation->block_holder);
        } 

    }

    return STATUS_OK;
}

__global__ void calculate_acceleration_values_block_reduce(double* d_masses, double* d_holder_position, double* d_aceleration, double* d_block_holder, int n, double d_dt, unsigned int block_n)
{   
    //Array where all values of this block will be stored
    extern __shared__ double sdata[];

    //Get position in the block
    int x = threadIdx.x;
    int y = threadIdx.y;

    //Get universal position
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    int j = threadIdx.y + blockIdx.y * blockDim.y;

    int ioffset = i * 3;
    int joffset = j * 3;

    //Dont calculate the pull of one body to itself
    if(i == j)
    {
        S(blockDim.x, blockDim.y, 0, x, y, sdata) = 0.0;    //x
        S(blockDim.x, blockDim.y, 1, x, y, sdata) = 0.0;    //y
        S(blockDim.x, blockDim.y, 2, x, y, sdata) = 0.0;    //z
    }
    //Calculate pull of one body by other body
    else if(i < n && j < n)
    {   
        double dx = d_holder_position[joffset] - d_holder_position[ioffset]; //rx body 2 - rx body 1
        double dy = d_holder_position[joffset+1] - d_holder_position[ioffset+1]; //ry body 2 - ry body 1
        double dz = d_holder_position[joffset+2] - d_holder_position[ioffset+2]; //rz body 2 - rz body 1
        
        double r = pow(dx, 2) + pow(dy, 2) + pow(dz, 2) + pow(softening, 2); //distance magnitud with some softening
        double h = ((G * d_masses[j]) / (pow(r, 1.5))); //Acceleration formula


        S(blockDim.x, blockDim.y, 0, x, y, sdata) = d_dt * h * dx; //Acceleration formula for x
        S(blockDim.x, blockDim.y, 1, x, y, sdata) = d_dt * h * dy; //Acceleration formula for y
        S(blockDim.x, blockDim.y, 2, x, y, sdata) = d_dt * h * dz; //Acceleration formula for z
    }
    
    //Reduce all values of this block
    if(i < n && j < n)
    {   
        __syncthreads();
        for (int thread = 0; thread < blockDim.y ; thread++)
        {
            if(y == thread)
            {
                B(n, block_n, 0, i, blockIdx.y, d_block_holder) += S(blockDim.x, blockDim.y, 0, x, y, sdata);
                B(n, block_n, 1, i, blockIdx.y, d_block_holder) += S(blockDim.x, blockDim.y, 1, x, y, sdata);
                B(n, block_n, 2, i, blockIdx.y, d_block_holder) += S(blockDim.x, blockDim.y, 2, x, y, sdata);
            }
            __syncthreads();
        }
    }
    /*if(!(blockIdx.y == block_n - 1) && 0==1)
    {   
        //Reduciton for 32 elements
        extern __shared__ int sdata[3][32];
        unsigned int tid = threadIdx.y;

        // each thread loads one element from global to shared mem
        sdata[0][tid] = D(n, 0, i, j, d_aceleration);
        sdata[1][tid] = D(n, 1, i, j, d_aceleration);
        sdata[2][tid] = D(n, 2, i, j, d_aceleration);
        __syncthreads();

        // do reduction in shared mem
        for (unsigned int s=1; s < blockDim.y; s *= 2) 
        {
            for (unsigned int s=blockDim.y/2; s>0; s>>=1) 
            {
                if (tid < s) 
                {
                    sdata[0][tid] += sdata[0][tid + s];
                    sdata[1][tid] += sdata[1][tid + s];
                    sdata[2][tid] += sdata[2][tid + s];
                }
                __syncthreads();
            }
        }
        // write result for this block to global mem
        if (tid == 0) 
        {
            B(n, block_n, 0, i,blockIdx.y, d_block_holder) = sdata[0][0];
            B(n, block_n, 1, i,blockIdx.y, d_block_holder) = sdata[1][0];
            B(n, block_n, 2, i,blockIdx.y, d_block_holder) = sdata[2][0];
        }
        

    }*/

    

}
